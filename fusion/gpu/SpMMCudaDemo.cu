#include "hip/hip_runtime.h"
//
// Created by mehdi on 6/16/24.
//
#include "aggregation/def.h"
#include "Timer.h"
#include <hipsparse.h>
#include <iostream>

#define WARMUP_NUM_CUDA 20
#define EXE_NUM_CUDA 200


void cusparseCSRSpMM(const sym_lib::CSR *mat1, const float *B,const int DenseRows, const int DenseCols, float* C, int algid, float& time) {
  swiftware::benchmark::Timer timer;
  const int n_rows_mat1 = mat1->m;
  const int n_cols_mat1 = mat1->n;
  const int n_rows_mat2 = DenseRows;
  const int n_cols_mat2 = DenseCols;
  const int m = n_rows_mat1;
  const int k = n_cols_mat1;
  const int n = n_cols_mat2;
  using scalar_t = float;
  scalar_t alpha = 1.0;
  scalar_t beta = 0.0;
  int nnzA = mat1->nnz;
  int *rowindA_csr = mat1->p;
  int *colindA = mat1->i;
  float* valuesA = new float[nnzA]; // TODO: Delete this allocated Arr
  for (int i=0; i < nnzA; i++){
    valuesA[i] = (float)mat1->x[i];
  }
  // cuda handle
  hipsparseHandle_t cusparse_handle = 0;
  hipsparseCreate(&cusparse_handle);
#if CUDART_VERSION < 11000
  int ldb = n;
  int ldc = m;
  hipsparseMatDescr_t descr = 0;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
  // kernel
  for (int i = 0; i < WARMUP_NUM_CUDA; i++) {
    hipsparseScsrmm2(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    HIPSPARSE_OPERATION_TRANSPOSE, m, n, k,
                    nnzA, &alpha, descr, valuesA, rowindA_csr, colindA,
                    B, ldb, &beta, C, ldc);
  }
  timer.Start();
  for (int i = 0; i < EXE_NUM_CUDA; i++) {
    hipsparseScsrmm2(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    HIPSPARSE_OPERATION_TRANSPOSE, m, n, k,
                    nnzA, &alpha, descr, valuesA, rowindA_csr, colindA,
                    B, ldb, &beta, C, ldc);
  }
  timer.Stop();
  time =  (float)timer.Elapsed()/EXE_NUM_CUDA;
  std::cout << "cuSPARSE 101 time = " << time << " ms" << std::endl;
#else
  hipsparseSpMatDescr_t matA;
  hipsparseDnMatDescr_t matB, matC;
  hipsparseCreateCsr(&matA,
                    m, k, nnzA,
                    rowindA_csr,
                    colindA,
                    valuesA,
                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
  hipsparseCreateDnMat(&matB,
                      k, n, n, (void *)B, HIP_R_32F, HIPSPARSE_ORDER_ROW);
  hipsparseCreateDnMat(&matC,
                      m, n, n,
                      C, HIP_R_32F, HIPSPARSE_ORDER_ROW);

  auto transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  auto transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  size_t workspace_size;
  hipsparseSpMMAlg_t alg = HIPSPARSE_SPMM_CSR_ALG2;
  if(algid == -1){
    alg = HIPSPARSE_SPMM_ALG_DEFAULT;
  } else if(algid == 2){
    alg = HIPSPARSE_SPMM_CSR_ALG2;
  } else if (algid == 3) {
    alg = HIPSPARSE_SPMM_CSR_ALG3;
  }
  for (int i = 0; i < WARMUP_NUM_CUDA; i++) {
    hipsparseSpMM_bufferSize(
        cusparse_handle, transA, transB,
        &alpha, matA, matB, &beta, matC,
        HIP_R_32F, alg,
        &workspace_size);
  }
  timer.startGPU();
  for (int i = 0; i < EXE_NUM_CUDA; i++) {
    hipsparseSpMM_bufferSize(
        cusparse_handle, transA, transB,
        &alpha, matA, matB, &beta, matC,
        HIP_R_32F, alg,
        &workspace_size);
  }
  timer.stopGPU("CuSparse CSR Buffer Time");
  std::cout << "cusparse csr buffer time: " <<  timer.ElapsedSeconds.count() / EXE_NUM_CUDA << " ms " << std::endl;
  void* workspace=NULL;
  hipMalloc(&workspace, workspace_size);
  for (int i = 0; i < WARMUP_NUM_CUDA; i++) {
    hipsparseSpMM(
        cusparse_handle, transA, transB,
        &alpha, matA, matB, &beta, matC,
        HIP_R_32F,alg,
        workspace);
  }
  swiftware::benchmark::Timer exeTime;
  timer.startGPU();
  for (int i = 0; i < EXE_NUM_CUDA; i++) {
    hipsparseSpMM(
        cusparse_handle, transA, transB,
        &alpha, matA, matB, &beta, matC,
        HIP_R_32F, alg,
        workspace);
  }
  timer.stopGPU("CuSparse CSR Exe Time");
  time = timer.ElapsedSeconds.count() / EXE_NUM_CUDA;
  std::cout << "cusparse csr exe time: " <<  time << " ms " << std::endl;
  hipFree(workspace);
  hipsparseDestroySpMat(matA);
  hipsparseDestroyDnMat(matB);
  hipsparseDestroyDnMat(matC);
#endif
}

int main () {
  int N = 1 << 20;
  float *x, *y, *d_x, *d_y;
  x = (float*) malloc(N*sizeof(float));
  y = (float*) malloc(N*sizeof(float));
  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++){
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = std::max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}