//
// Created by salehm32 on 20/06/24.
//
#include "../Cuda_SpMM_SpMM_Demo_Utils.h"

#include <iostream>

using namespace sym_lib;
int main (const int argc, const char *argv[]) {
  TestParameters tp;
  tp._order_method = SYM_ORDERING::NONE;
  ScheduleParameters sp;
  Stats *stats;
  parse_args(argc, argv, &sp, &tp);
  CSC *aCSC = get_matrix_from_parameter(&tp);
  if (aCSC->m != aCSC->n) {
    return -1;
  }
  CSC *aCSCFull = nullptr;
  if (aCSC->stype == -1 || aCSC->stype == 1) {
    aCSCFull = make_full(aCSC);
  } else {
    aCSCFull = copy_sparse(aCSC);
  }
  tp._dim1 = aCSCFull->m;
  tp._dim2 = aCSCFull->n;
  tp._nnz = aCSCFull->nnz;
  tp._density = (double)tp._nnz / (double)(tp._dim1 * tp._dim2);

  int numThread = sp._num_threads, numTrial = 5;
  int ufThreadsPerBlock = 128;
  int fThreadsPerBlock=32;
  std::string expName = "SpMM_SpMM_Demo";
  auto *inSpMM =
      new CudaTensorInputs(aCSCFull->m, tp._b_cols, aCSCFull->m, aCSCFull->m,
                           aCSCFull, aCSCFull, numThread, numTrial, expName);

  stats = new swiftware::benchmark::Stats("CPU_Unfused_Seq", "SpMMSpMM", numTrial,
                                          tp._matrix_name, numThread);
  auto *cpuSpMMSpMM = new SeqSpMMSpMM(inSpMM, stats);
  cpuSpMMSpMM->run();
//  std::cout << "CPU: " << std::endl;
//  cpuSpMMSpMM->OutTensor->printDx();
  std::copy(cpuSpMMSpMM->OutTensor->Xx,
            cpuSpMMSpMM->OutTensor->Xx +
                cpuSpMMSpMM->OutTensor->M * cpuSpMMSpMM->OutTensor->N,
            inSpMM->CorrectSol);
  inSpMM->IsSolProvided = true;
  auto headerStat = cpuSpMMSpMM->printStatsHeader();
  auto cpuSpMMSpMMStat = cpuSpMMSpMM->printStats();
  delete cpuSpMMSpMM;
  delete stats;

  stats = new swiftware::benchmark::Stats("GPU_Unfused_CuSparse_ALG2","SpMMSpMM", numTrial,tp._matrix_name,numThread);
  auto *unfusedCuSparseAlg2 = new SpMMSpMMCuSparse(inSpMM,stats, HIPSPARSE_SPMM_CSR_ALG2);
  unfusedCuSparseAlg2->run();
  //  std::cout << "UNFUSED: " << std::endl;
  //  unfusedCuSparseAlg2->OutTensor->printDx();
  auto unfusedCuSparseAlg2Stat = unfusedCuSparseAlg2->printStats();
  delete unfusedCuSparseAlg2;
  delete stats;

  stats = new swiftware::benchmark::Stats("GPU_Unfused_CuSparse_ALG3","SpMMSpMM", numTrial,tp._matrix_name,numThread);
  auto *unfusedCuSparseAlg3 = new SpMMSpMMCuSparse(inSpMM,stats, HIPSPARSE_SPMM_CSR_ALG3);
  unfusedCuSparseAlg3->run();
  //  std::cout << "UNFUSED: " << std::endl;
  //  unfusedCuSparseAlg3->OutTensor->printDx();
  auto unfusedCuSparseAlg3Stat = unfusedCuSparseAlg3->printStats();
  delete unfusedCuSparseAlg3;
  delete stats;

  stats = new swiftware::benchmark::Stats("GPU_Unfused_SeqReduceRowBalance","SpMMSpMM", numTrial,tp._matrix_name,numThread);
  auto *unfusedSeqReduceRowBalance = new SpMMSpMMSeqReduceRowBalance(inSpMM,stats, ufThreadsPerBlock);
  unfusedSeqReduceRowBalance->run();
  //  std::cout << "UNFUSED: " << std::endl;
  //  unfusedSeqReduceRowBalance->OutTensor->printDx();
  auto unfusedSeqReduceRowBalanceStat = unfusedSeqReduceRowBalance->printStats();
  delete unfusedSeqReduceRowBalance;
  delete stats;

//  stats = new swiftware::benchmark::Stats("GPU_Fused_SeqReduceRowBalance","SpMMSpMM", numTrial,tp._matrix_name,numThread);
//  auto *fusedSeqReduceRowBalance = new FusedSpMMSpMMSeqReduceRowBalance(inSpMM,stats, ThreadsPerBlock);
//  fusedSeqReduceRowBalance->run();
////  std::cout << "FUSED: " << std::endl;
////  fusedSeqReduceRowBalance->OutTensor->printDx();
//  auto fusedSeqReduceRowBalanceStat = fusedSeqReduceRowBalance->printStats();
//  delete fusedSeqReduceRowBalance;
//  delete stats;
//
//  stats = new swiftware::benchmark::Stats("GPU_Fused_Reordered_SeqReduceRowBalance","SpMMSpMM", numTrial,tp._matrix_name,numThread);
//  auto *fusedSeqReduceRowBalanceReordered = new FusedSpMMSpMMSeqReduceRowBalanceReordered(inSpMM,stats, ThreadsPerBlock);
//  fusedSeqReduceRowBalanceReordered->run();
//  //  std::cout << "FUSED: " << std::endl;
//  //  fusedSeqReduceRowBalanceReordered->OutTensor->printDx();
//  auto fusedSeqReduceRowBalanceReorderedStat = fusedSeqReduceRowBalanceReordered->printStats();
//  delete fusedSeqReduceRowBalanceReordered;
//  delete stats;

  stats = new swiftware::benchmark::Stats("GPU_Fused_Reordered_HighFusionRatio_8","SpMMSpMM", numTrial,tp._matrix_name,numThread);
  auto *fusedHighFusionRatio8 = new FusedSpMMSpMMHighFusionRatio(inSpMM,stats, ufThreadsPerBlock, ufThreadsPerBlock, 8);
  fusedHighFusionRatio8->run();
  //  std::cout << "FUSED: " << std::endl;
  //  fusedHighFusionRatio8->OutTensor->printDx();
  auto fusedHighFusionRatio8Stat = fusedHighFusionRatio8->printStats();
  delete fusedHighFusionRatio8;
  delete stats;

  stats = new swiftware::benchmark::Stats("GPU_Fused_Reordered_HighFusionRatio_16","SpMMSpMM", numTrial,tp._matrix_name,numThread);
  auto *fusedHighFusionRatio16 = new FusedSpMMSpMMHighFusionRatio(inSpMM,stats, ufThreadsPerBlock, ufThreadsPerBlock, 16);
  fusedHighFusionRatio16->run();
  //  std::cout << "FUSED: " << std::endl;
  //  fusedHighFusionRatio16->OutTensor->printDx();
  auto fusedHighFusionRatio16Stat = fusedHighFusionRatio16->printStats();
  delete fusedHighFusionRatio16;
  delete stats;

  stats = new swiftware::benchmark::Stats("GPU_Fused_Reordered_HighFusionRatio_32","SpMMSpMM", numTrial,tp._matrix_name,numThread);
  auto *fusedHighFusionRatio32 = new FusedSpMMSpMMHighFusionRatio(inSpMM,stats, ufThreadsPerBlock, ufThreadsPerBlock, 32);
  fusedHighFusionRatio32->run();
  //  std::cout << "FUSED: " << std::endl;
  //  fusedHighFusionRatio32->OutTensor->printDx();
  auto fusedHighFusionRatio32Stat = fusedHighFusionRatio32->printStats();
  delete fusedHighFusionRatio32;
  delete stats;

  stats = new swiftware::benchmark::Stats("GPU_Fused_Reordered_HighFusionRatio_64","SpMMSpMM", numTrial,tp._matrix_name,numThread);
  auto *fusedHighFusionRatio64 = new FusedSpMMSpMMHighFusionRatio(inSpMM,stats, ufThreadsPerBlock, ufThreadsPerBlock, 64);
  fusedHighFusionRatio64->run();
  //  std::cout << "FUSED: " << std::endl;
  //  fusedHighFusionRatio64->OutTensor->printDx();
  auto fusedHighFusionRatio64Stat = fusedHighFusionRatio64->printStats();
  delete fusedHighFusionRatio64;
  delete stats;


  stats = new swiftware::benchmark::Stats("GPU_Fused_Reordered_32_HighFusionRatio_8","SpMMSpMM", numTrial,tp._matrix_name,numThread);
  auto *fusedHighFusionRatio_32_8 = new FusedSpMMSpMMHighFusionRatio(inSpMM,stats, 32, ufThreadsPerBlock, 8);
  fusedHighFusionRatio_32_8->run();
  //  std::cout << "FUSED: " << std::endl;
  //  fusedHighFusionRatio_32_8->OutTensor->printDx();
  auto fusedHighFusionRatio_32_8Stat = fusedHighFusionRatio_32_8->printStats();
  delete fusedHighFusionRatio_32_8;
  delete stats;

  stats = new swiftware::benchmark::Stats("GPU_Fused_Reordered_32_HighFusionRatio_16","SpMMSpMM", numTrial,tp._matrix_name,numThread);
  auto *fusedHighFusionRatio_32_16 = new FusedSpMMSpMMHighFusionRatio(inSpMM,stats, 32, ufThreadsPerBlock, 16);
  fusedHighFusionRatio_32_16->run();
  //  std::cout << "FUSED: " << std::endl;
  //  fusedHighFusionRatio_32_16->OutTensor->printDx();
  auto fusedHighFusionRatio_32_16Stat = fusedHighFusionRatio_32_16->printStats();
  delete fusedHighFusionRatio_32_16;
  delete stats;

  stats = new swiftware::benchmark::Stats("GPU_Fused_Reordered_32_HighFusionRatio_32","SpMMSpMM", numTrial,tp._matrix_name,numThread);
  auto *fusedHighFusionRatio_32_32 = new FusedSpMMSpMMHighFusionRatio(inSpMM,stats, 32, ufThreadsPerBlock, 32);
  fusedHighFusionRatio_32_32->run();
  //  std::cout << "FUSED: " << std::endl;
  //  fusedHighFusionRatio_32_32->OutTensor->printDx();
  auto fusedHighFusionRatio_32_32Stat = fusedHighFusionRatio_32_32->printStats();
  delete fusedHighFusionRatio_32_32;
  delete stats;

  stats = new swiftware::benchmark::Stats("GPU_Fused_Reordered_32_HighFusionRatio_64","SpMMSpMM", numTrial,tp._matrix_name,numThread);
  auto *fusedHighFusionRatio_32_64 = new FusedSpMMSpMMHighFusionRatio(inSpMM,stats, 32, ufThreadsPerBlock, 64);
  fusedHighFusionRatio_32_64->run();
  //  std::cout << "FUSED: " << std::endl;
  //  fusedHighFusionRatio_32_64->OutTensor->printDx();
  auto fusedHighFusionRatio_32_64Stat = fusedHighFusionRatio_32_64->printStats();
  delete fusedHighFusionRatio_32_64;
  delete stats;

//  stats = new swiftware::benchmark::Stats("GPU_Fused_SeqReduceBColsBlocking_WSM","SpMMSpMM", numTrial,tp._matrix_name,numThread);
//  auto *fusedSeqReduceWSM = new FusedSpMMSpMMSeqReduceBColsBlockingWithSharedMem(inSpMM, stats, ThreadsPerBlock, tp._b_cols);
//  fusedSeqReduceWSM->run();
//  //  std::cout << "FUSED: " << std::endl;
//  //  fusedSeqReduceBColsBlockinWSMg->OutTensor->printDx();
//  auto fusedSeqReduceWSMStat = fusedSeqReduceWSM->printStats();
//  delete fusedSeqReduceWSM;
//  delete stats;
//
//  stats = new swiftware::benchmark::Stats("GPU_Fused_SeqReduceBColsBlocking_4","SpMMSpMM", numTrial,tp._matrix_name,numThread);
//  auto *fusedSeqReduceBColsBlocking4 = new FusedSpMMSpMMSeqReduceBColsBlocking(inSpMM, stats, ThreadsPerBlock, 4);
//  fusedSeqReduceBColsBlocking4->run();
//  //  std::cout << "FUSED: " << std::endl;
//  //  fusedSeqReduceBColsBlocking->OutTensor->printDx();
//  auto fusedSeqReduceBColsBlocking4Stat = fusedSeqReduceBColsBlocking4->printStats();
//  delete fusedSeqReduceBColsBlocking4;
//  delete stats;
//
//  stats = new swiftware::benchmark::Stats("GPU_Fused_SeqReduceBColsBlocking_WSM_4","SpMMSpMM", numTrial,tp._matrix_name,numThread);
//  auto *fusedSeqReduceBColsBlockingWSM4 = new FusedSpMMSpMMSeqReduceBColsBlockingWithSharedMem(inSpMM, stats, ThreadsPerBlock, 4);
//  fusedSeqReduceBColsBlockingWSM4->run();
//  //  std::cout << "FUSED: " << std::endl;
//  //  fusedSeqReduceBColsBlockinWSMg->OutTensor->printDx();
//  auto fusedSeqReduceBColsBlockingWSM4Stat = fusedSeqReduceBColsBlockingWSM4->printStats();
//  delete fusedSeqReduceBColsBlockingWSM4;
//  delete stats;
//
//  stats = new swiftware::benchmark::Stats("GPU_Fused_SeqReduceBColsBlocking_8","SpMMSpMM", numTrial,tp._matrix_name,numThread);
//  auto *fusedSeqReduceBColsBlocking8 = new FusedSpMMSpMMSeqReduceBColsBlocking(inSpMM, stats, ThreadsPerBlock, 8);
//  fusedSeqReduceBColsBlocking8->run();
//  //  std::cout << "FUSED: " << std::endl;
//  //  fusedSeqReduceBColsBlocking->OutTensor->printDx();
//  auto fusedSeqReduceBColsBlocking8Stat = fusedSeqReduceBColsBlocking8->printStats();
//  delete fusedSeqReduceBColsBlocking8;
//  delete stats;
//
//  stats = new swiftware::benchmark::Stats("GPU_Fused_SeqReduceBColsBlocking_WSM_8","SpMMSpMM", numTrial,tp._matrix_name,numThread);
//  auto *fusedSeqReduceBColsBlockingWSM8 = new FusedSpMMSpMMSeqReduceBColsBlockingWithSharedMem(inSpMM, stats, ThreadsPerBlock, 8);
//  fusedSeqReduceBColsBlockingWSM8->run();
//  //  std::cout << "FUSED: " << std::endl;
//  //  fusedSeqReduceBColsBlockinWSMg->OutTensor->printDx();
//  auto fusedSeqReduceBColsBlockingWSM8Stat = fusedSeqReduceBColsBlockingWSM8->printStats();
//  delete fusedSeqReduceBColsBlockingWSM8;
//  delete stats;
//
//  stats = new swiftware::benchmark::Stats("GPU_Fused_SeqReduceBColsBlocking_16","SpMMSpMM", numTrial,tp._matrix_name,numThread);
//  auto *fusedSeqReduceBColsBlocking16 = new FusedSpMMSpMMSeqReduceBColsBlocking(inSpMM, stats, ThreadsPerBlock, 16);
//  fusedSeqReduceBColsBlocking16->run();
//  //  std::cout << "FUSED: " << std::endl;
//  //  fusedSeqReduceBColsBlocking->OutTensor->printDx();
//  auto fusedSeqReduceBColsBlocking16Stat = fusedSeqReduceBColsBlocking16->printStats();
//  delete fusedSeqReduceBColsBlocking16;
//  delete stats;
//
//  stats = new swiftware::benchmark::Stats("GPU_Fused_SeqReduceBColsBlocking_WSM_16","SpMMSpMM", numTrial,tp._matrix_name,numThread);
//  auto *fusedSeqReduceBColsBlockingWSM16 = new FusedSpMMSpMMSeqReduceBColsBlockingWithSharedMem(inSpMM, stats, ThreadsPerBlock, 16);
//  fusedSeqReduceBColsBlockingWSM16->run();
//  //  std::cout << "FUSED: " << std::endl;
//  //  fusedSeqReduceBColsBlockinWSMg->OutTensor->printDx();
//  auto fusedSeqReduceBColsBlockingWSM16Stat = fusedSeqReduceBColsBlockingWSM16->printStats();
//  delete fusedSeqReduceBColsBlockingWSM16;
//  delete stats;
//
//  stats = new swiftware::benchmark::Stats("GPU_Fused_SeqReduceBColsBlocking_32","SpMMSpMM", numTrial,tp._matrix_name,numThread);
//  auto *fusedSeqReduceBColsBlocking32 = new FusedSpMMSpMMSeqReduceBColsBlocking(inSpMM, stats, ThreadsPerBlock, 32);
//  fusedSeqReduceBColsBlocking32->run();
//  //  std::cout << "FUSED: " << std::endl;
//  //  fusedSeqReduceBColsBlocking->OutTensor->printDx();
//  auto fusedSeqReduceBColsBlocking32Stat = fusedSeqReduceBColsBlocking32->printStats();
//  delete fusedSeqReduceBColsBlocking32;
//  delete stats;
//
//  stats = new swiftware::benchmark::Stats("GPU_Fused_SeqReduceBColsBlocking_WSM_32","SpMMSpMM", numTrial,tp._matrix_name,numThread);
//  auto *fusedSeqReduceBColsBlockingWSM32 = new FusedSpMMSpMMSeqReduceBColsBlockingWithSharedMem(inSpMM, stats, ThreadsPerBlock, 32);
//  fusedSeqReduceBColsBlockingWSM32->run();
//  //  std::cout << "FUSED: " << std::endl;
//  //  fusedSeqReduceBColsBlockinWSMg->OutTensor->printDx();
//  auto fusedSeqReduceBColsBlockingWSM32Stat = fusedSeqReduceBColsBlockingWSM32->printStats();
//  delete fusedSeqReduceBColsBlockingWSM32;
//  delete stats;

//  stats = new swiftware::benchmark::Stats("GPU_Fused_SeqReduceBColsBlocking_64","SpMMSpMM", numTrial,tp._matrix_name,numThread);
//  auto *fusedSeqReduceBColsBlocking64 = new FusedSpMMSpMMSeqReduceBColsBlocking(inSpMM, stats, ThreadsPerBlock, 64);
//  fusedSeqReduceBColsBlocking64->run();
//  //  std::cout << "FUSED: " << std::endl;
//  //  fusedSeqReduceBColsBlocking->OutTensor->printDx();
//  auto fusedSeqReduceBColsBlocking64Stat = fusedSeqReduceBColsBlocking64->printStats();
//  delete fusedSeqReduceBColsBlocking64;
//  delete stats;
//
//  stats = new swiftware::benchmark::Stats("GPU_Fused_SeqReduceBColsBlocking_128","SpMMSpMM", numTrial,tp._matrix_name,numThread);
//  auto *fusedSeqReduceBColsBlocking128 = new FusedSpMMSpMMSeqReduceBColsBlocking(inSpMM,stats, ThreadsPerBlock, 128);
//  fusedSeqReduceBColsBlocking128->run();
//  //  std::cout << "FUSED: " << std::endl;
//  //  fusedSeqReduceBColsBlocking->OutTensor->printDx();
//  auto fusedSeqReduceBColsBlocking128Stat = fusedSeqReduceBColsBlocking128->printStats();
//  delete fusedSeqReduceBColsBlocking128;
//  delete stats;

  std::string profHeader = "";
  std::string profStat = "";

  auto csvInfo = sp.print_csv(true);
  std::string spHeader = std::get<0>(csvInfo);
  std::string spStat = std::get<1>(csvInfo);

  auto tpCsv = tp.print_csv(true);
  std::string tpHeader = std::get<0>(tpCsv);
  std::string tpStat = std::get<1>(tpCsv);




  if (tp.print_header)
    std::cout << headerStat + spHeader + tpHeader + profHeader << std::endl;
  std::cout << cpuSpMMSpMMStat << spStat + tpStat + profStat << std::endl;
  std::cout << unfusedSeqReduceRowBalanceStat << spStat + tpStat + profStat << std::endl;
//  std::cout << unfusedCuSparseAlg2Stat << spStat + tpStat + profStat << std::endl;
//  std::cout << unfusedCuSparseAlg3Stat << spStat + tpStat + profStat << std::endl;
//  std::cout << fusedSeqReduceRowBalanceStat << spStat + tpStat + profStat << std::endl;
//  std::cout << fusedSeqReduceRowBalanceReorderedStat << spStat + tpStat + profStat << std::endl;
  std::cout << fusedHighFusionRatio8Stat << spStat + tpStat + profStat << std::endl;
  std::cout << fusedHighFusionRatio16Stat << spStat + tpStat + profStat << std::endl;
  std::cout << fusedHighFusionRatio32Stat << spStat + tpStat + profStat << std::endl;
  std::cout << fusedHighFusionRatio64Stat << spStat + tpStat + profStat << std::endl;
  std::cout << fusedHighFusionRatio_32_8Stat << spStat + tpStat + profStat << std::endl;
  std::cout << fusedHighFusionRatio_32_16Stat << spStat + tpStat + profStat << std::endl;
  std::cout << fusedHighFusionRatio_32_32Stat << spStat + tpStat + profStat << std::endl;
  std::cout << fusedHighFusionRatio_32_64Stat << spStat + tpStat + profStat << std::endl;
//  std::cout << fusedSeqReduceWSMStat << spStat + tpStat + profStat << std::endl;
//  std::cout << fusedSeqReduceBColsBlocking4Stat << spStat + tpStat + profStat << std::endl;
//  std::cout << fusedSeqReduceBColsBlockingWSM4Stat << spStat + tpStat + profStat << std::endl;
//  std::cout << fusedSeqReduceBColsBlocking8Stat << spStat + tpStat + profStat << std::endl;
//  std::cout << fusedSeqReduceBColsBlockingWSM8Stat << spStat + tpStat + profStat << std::endl;
//  std::cout << fusedSeqReduceBColsBlocking16Stat << spStat + tpStat + profStat << std::endl;
//  std::cout << fusedSeqReduceBColsBlockingWSM16Stat << spStat + tpStat + profStat << std::endl;
//  std::cout << fusedSeqReduceBColsBlocking32Stat << spStat + tpStat + profStat << std::endl;
//  std::cout << fusedSeqReduceBColsBlockingWSM32Stat << spStat + tpStat + profStat << std::endl;
//  std::cout << fusedSeqReduceBColsBlocking64Stat << spStat + tpStat + profStat << std::endl;
//  std::cout << fusedSeqReduceBColsBlocking128Stat << spStat + tpStat + profStat << std::endl;


  delete inSpMM;

}